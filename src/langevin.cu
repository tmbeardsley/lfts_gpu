#include "hip/hip_runtime.h"
// #############################################################################
// Performs a langevin update of w-(r) and keeps track of symmetrised noise
// #############################################################################

#include "langevin.h"

// Langevin update of w-(r) on the GPU using symmetrised noise
static __global__ void langevin_sym(double *w_gpu, double *noise_gpu_new, double *noise_gpu_prev, const double XbN, const double dt, const int M)
{
	int const tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= M) return;
	w_gpu[tid] += -(w_gpu[tid+2*M]+2*w_gpu[tid]/XbN)*dt+0.5*(noise_gpu_prev[tid]+noise_gpu_new[tid]);
}



langevin::langevin(hiprandGenerator_t &RNG, double sigma, int M, int TpB) {
    TpB_ = TpB;
    M_ = M;

    // Allocate memory for Gaussian random noise on the GPU
    GPU_ERR(hipMalloc((void**)&noise_gpu_,2*M_*sizeof(double)));

    // Generate initial "previous" Gaussian random noise on the gpu
    hiprandGenerateNormalDouble(RNG, noise_gpu_, M_, 0.0, sigma);
    noise_gpu_prev_ = noise_gpu_;
    noise_gpu_new_ = noise_gpu_ + M_;
}

langevin::~langevin() {
    GPU_ERR(hipFree(noise_gpu_));
}

// Perform a Langevin update of the fields using symmetrised noise
void langevin::step_wm(double* w_gpu, hiprandGenerator_t &RNG, double XbN, double sigma, double dt)
{
    double *ptr_tmp;

    // Create new random noise on the GPU for the call to langevin()
    hiprandGenerateNormalDouble(RNG, noise_gpu_new_, M_, 0.0, sigma);

    // Perform the Langevin step on the GPU
    langevin_sym<<<(M_+TpB_-1)/TpB_,TpB_>>>(w_gpu, noise_gpu_new_, noise_gpu_prev_, XbN, dt, M_);

    // Swap noise pointer positions to avoid shifting (copying) data every step
    ptr_tmp = noise_gpu_prev_;
    noise_gpu_prev_ = noise_gpu_new_;
    noise_gpu_new_ = ptr_tmp;
}

