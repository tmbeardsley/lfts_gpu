// #####################################################################
// Functions to manage problems arising from CUDA and the cuFFT library
// #####################################################################

#include "GPUerror.h"


void HandleError(hipError_t err, char const * const file, int const line)
{
    if (err != hipSuccess) {
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

void HandleError(hipfftResult err, char const * const file, int const line)
{
    if (err != HIPFFT_SUCCESS) {
        switch(err) {
            case HIPFFT_INVALID_PLAN:
                printf ("cufft %s in %s at line %d\n", "HIPFFT_INVALID_PLAN", file, line);
                break;
            case HIPFFT_INVALID_VALUE:
                printf ("cufft %s in %s at line %d\n", "HIPFFT_INVALID_VALUE", file, line);
                break;
            case HIPFFT_INTERNAL_ERROR:
                printf ("cufft %s in %s at line %d\n", "HIPFFT_INTERNAL_ERROR", file, line);
                break;
            case HIPFFT_EXEC_FAILED:
                printf ("cufft %s in %s at line %d\n", "HIPFFT_EXEC_FAILED", file, line);
                break;
            case HIPFFT_SETUP_FAILED:
                printf ("cufft %s in %s at line %d\n", "CUFFT_EXEC_FAILEDCUFFT_SETUP_FAILED", file, line);
                break;
            default:
                printf ("cufft %s in %s at line %d\n", "CUFFT_EXEC_FAILEDCUFFT_SETUP_FAILED", file, line);
        }
        exit(EXIT_FAILURE);
    }
}


