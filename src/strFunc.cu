#include "hip/hip_runtime.h"
// #####################################################################################
// Provides the public methods: void sample(...) and void save(...), 
// which take samples of the structure funtion, S(k), and save the spherically-averaged 
// S(k) to file.
// S(k) should only be calculated in simulations keeping L[] and XbN constant.
// #####################################################################################

#include "strFunc.h"


// Multiply and sum propagators for calculating either phiA[r] or phiB[r]
static __global__ void add_norm(double *S_gpu, hipfftDoubleComplex *wk_gpu, const int Mk)
{
	int const tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= Mk) return;
	S_gpu[tid] += pow(wk_gpu[tid].x, 2.0) + pow(wk_gpu[tid].y, 2.0);
}


// Constructor
strFunc::strFunc(int *m, double *L, int M, int Mk, double CV, double chi_b, int TpB, double dK) :
    TpB_(TpB),
    S_gpu_(create_unique_cuda_memory<double>(Mk)),
    K_(std::make_unique<double[]>(Mk)),
    dK_(dK),
    coeff_(CV/(chi_b*chi_b*M*M)),
    wt_(std::make_unique<int[]>(Mk)),
    P_(std::make_unique<int[]>(Mk)),
    nsamples_(0),
    wk_(std::make_unique<std::complex<double>[]>(Mk)),
    wk_gpu_(create_unique_cuda_memory<hipfftDoubleComplex>(Mk)),
    chi_b_(chi_b),
    Mk_(Mk),
    wr_to_wk_(new hipfftHandle, cufftDeleter())
    //wr_to_wk_(std::make_unique<>())
{

    // Allocate memory for S(k) on the GPU
    Array_init<<<(Mk_+TpB_-1)/TpB_, TpB_>>>(S_gpu_.get(), 0.0, Mk);

    // Create a cufft plan for the Fourier transform on the GPU
    GPU_ERR(hipfftPlan3d(wr_to_wk_.get(), m[0], m[1], m[2], HIPFFT_D2Z));

    // Populate the wavevector modulus array, K_
    calcK(K_.get(), m, L);

    // Populate the map, P_, which puts the wavevector moduli, K_, into ascending order
    std::iota(P_.get(), P_.get() + Mk_, 0);
    std::stable_sort(P_.get(), P_.get()+Mk_, [this](size_t i, size_t j) {return K_[i] < K_[j];});
}


// Sample norm(w-(k)) 
void strFunc::sample(double *w_gpu) {
    // Transform w-(r) to k-space to get w-(k)
    GPU_ERR(hipfftExecD2Z(*(wr_to_wk_.get()), w_gpu, wk_gpu_.get()));

    // Sample the norm of w-(k) for each wavevector and add to its sum
    add_norm<<<(Mk_+TpB_-1)/TpB_, TpB_>>>(S_gpu_.get(), wk_gpu_.get(), Mk_);

    // Increment the number of samples
    nsamples_++;
}


// Output the spherically-averaged structure function to file
void strFunc::save(std::string fileName, int dp) {
    double S_sum = 0.0, *S;
    int k, n_same = 0;
    std::ofstream out_stream;

    out_stream.open(fileName);
    out_stream.precision(dp);
    out_stream.setf(std::ios::fixed, std::ios::floatfield);

    // Copy S_gpu to the host
    S = new double[Mk_];
    GPU_ERR(hipMemcpy(S, S_gpu_.get(), Mk_*sizeof(double), hipMemcpyDeviceToHost));

    // Spherical average of S(k)
    for (k=0; k<Mk_; k++) {
        // Take into account vector weighting from the FFT and sum S for repeated K-vectors
        S_sum += wt_[P_[k]] * ((coeff_/nsamples_)*S[P_[k]] - 0.5/chi_b_);
        n_same += wt_[P_[k]];

        // Output value for current K-vector when difference in K exceeds tolerence dK_
        if ( (k==Mk_-1) || (fabs(K_[P_[k+1]]-K_[P_[k]]) > dK_) ) {
            out_stream << K_[P_[k]] << "\t" << S_sum/n_same << std::endl;

            // Reset summations for next K-vector
            S_sum = 0.0;
            n_same = 0;
        }
    } 
    out_stream.close();

    delete[] S;
}


// Destructor
strFunc::~strFunc() {
    //GPU_ERR(hipfftDestroy(wr_to_wk_));
}


// Calculate the wavevector moduli and store in K[]
void strFunc::calcK(double *K, int *_m, double *_L) {

    int K0, K1, k;
    double kx_sq, ky_sq, kz_sq;

    for (k=0; k<Mk_; k++) wt_[k]=2;

    for (int k0=-(_m[0]-1)/2; k0<=_m[0]/2; k0++) {
        K0 = (k0<0)?(k0+_m[0]):k0;
        kx_sq = k0*k0/(_L[0]*_L[0]);

        for (int k1=-(_m[1]-1)/2; k1<=_m[1]/2; k1++) {
            K1 = (k1<0)?(k1+_m[1]):k1;
            ky_sq = k1*k1/(_L[1]*_L[1]);

            for (int k2=0; k2<=_m[2]/2; k2++) {
                kz_sq = k2*k2/(_L[2]*_L[2]);
                k = k2 + (_m[2]/2+1)*(K1+_m[1]*K0);
                K[k] = 2*M_PI*pow(kx_sq+ky_sq+kz_sq,0.5); 
                if ((k2==0)||(k2==_m[2]/2)) wt_[k]=1;
            }
        }
    }
}


